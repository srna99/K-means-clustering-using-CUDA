#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <iostream>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"
#include <hip/hip_runtime.h>

using namespace std;


#define THREADS_PER_BLOCK 256

// Calculate distance from one datapoint to another
__device__ float CalculateDistance(float *a, float *b, int size) {
    float sum = 0;
    float diff;
    
    for(int i = 0; i < size; i++) {
        diff = a[i] - b[i];
        sum += diff * diff;
    }
    
    return sum;
}

// Assign each datapoint to nearest centroid
__global__ void AssignClusters(float *dataset, float *centroids, int *clusters, int k, int numInst, int numAttr) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < numInst) {
        float minDistance = FLT_MAX;

        for(int i = 0; i < k; i++) {
            float dist = CalculateDistance(&dataset[tid * numAttr], &centroids[i * numAttr], numAttr);

            if(dist < minDistance) {
                minDistance = dist;
                clusters[tid] = i;
            }
        }
    }
}

// Calculate the sums for each cluster and their total sizes
__global__ void CalculateClusterSumsAndSizes(float *dataset, int *clusters, float *sumOfCentroids, int *clusterSizes, int k, int numInst, int numAttr) {
    extern __shared__ int sharedMemory[];

    int *sharedClusters = sharedMemory;
    float *sharedDataset = (float *) &sharedMemory[THREADS_PER_BLOCK];

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int th_idx = threadIdx.x;
    
    if(tid < numInst) {
        for(int i = 0; i < numAttr; i++) {
            sharedDataset[th_idx + i] = dataset[tid * numAttr + i];
        }

        sharedClusters[th_idx] = clusterSizes[tid];
    } else {
        for(int i = 0; i < numAttr; i++) {
            sharedDataset[th_idx + i] = 0;
        }

        sharedClusters[th_idx] = -1;
    }

    __syncthreads();

    if(th_idx < numAttr) {
        float attrSum[k] = {0};

        for(int i = 0; i < THREADS_PER_BLOCK; i++) {
            if(sharedClusters[i] != -1) {
                attrSum[sharedClusters[i]] += sharedDataset[i * numAttr + th_idx];
            }
        }

        for(int i = 0; i < k; i++) {
            atomicAdd(&sumOfCentroids[i * numAttr + th_idx], attrSum[i]);
        }
    } else if(th_idx == numAttr + 1) {
        int clusterAmounts[k] = {0};

        for(int i = 0; i < THREADS_PER_BLOCK; i++) {
            if(sharedClusters[i] != -1) {
                clusterSize[sharedClusters[i]]++;
            }
        }

        for(int i = 0; i < k; i++) {
            atomicAdd(&clusterSizes[i], clusterAmounts[i]);
        }
    }

    // __syncthreads();

    // if(th_idx == 0) {
    //     int clusterAmounts[k] = {0};

    //     for(int i = 0; i < THREADS_PER_BLOCK; i++) {
    //         if(sharedClusters[i] != -1) {
    //             clusterSize[sharedClusters[i]]++;
    //         }
    //     }

    //     for(int i = 0; i < k; i++) {
    //         atomicAdd(&clusterSizes[i], clusterAmounts[i]);
    //     }
    // }
}

// Calculate means for each cluster for new centroids
__global__ void CalculateCentroidMeans(float *centroids, float *sumOfCentroids, int *clusterSizes, int k, int numAttr) {
    // for(int i = 0; i < k * numAttr; i++) {
    //     float newCentroid = sumOfValues[i] / clusterSize[(int) (i / numAttr)];

    //     centroidDiffs[i] = abs(centroids[i] - newCentroid);

    //     centroids[i] = newCentroid;
    //     printf("Centroid #%d: Attr%d = %f, Diff = %f, Cluster Size = %d\n", (int) (i / numAttr) + 1, i % numAttr, centroids[i], centroidDiffs[i], clusterSize[(int) (i / numAttr)]);
    // }
}

int main(int argc, char *argv[]) {
    if(argc != 3)
    {
        cout << "Usage: ./cuda-kmeans datasets/dataset.arff k" << endl;
        exit(0);
    }

    // Number of clusters
    int k = strtol(argv[2], NULL, 10);

    srand(13);

    // Parse dataset
    ArffParser parser(argv[1]);
    ArffData *dataset = parser.parse();

    int numAttr = dataset->num_attributes() - 1;
    int numInst = dataset->num_instances();

    // Allocate host memory
    float *h_dataset = (float *) malloc(numInst * numAttr * sizeof(float));
    float *h_centroids = (float *) malloc(k * numAttr * sizeof(float));
    int *h_clusters = (int *) malloc(numInst * sizeof(int));

    for(int i = 0; i < numInst; i++) {
        for(int j = 0; j < numAttr; j++) {
            h_dataset[i * numAttr + j] = dataset->get_instance(i)->get(j)->operator float();
        }
    }

    // Initialize centroids as random datapoints
    for(int i = 0; i < k; i++) {
        int randPoint = rand() % (numInst + 1);

        for(int j = 0; j < numAttr; j++) {
            h_centroids[i * numAttr + j] = h_dataset[randPoint * numAttr + j];
        }
    }

    // Allocate device memory
    float *d_dataset;
    float *d_centroids;
    int *d_clusters;
    float *d_sumOfCentroids;
    // float *d_centroidDiffs;
    int *d_clusterSizes;

    hipMalloc(&d_dataset, numInst * numAttr * sizeof(float));
    hipMalloc(&d_centroids, k * numAttr * sizeof(float));
    hipMalloc(&d_clusters, numInst * sizeof(int));
    hipMalloc(&d_sumOfCentroids, k * numAttr * sizeof(float));
    // hipMalloc(&d_centroidDiffs, k * num_attr, sizeof(float));
    hipMalloc(&d_clusterSizes, k * sizeof(int));

    hipMemset(d_sumOfValues, 0, k * numAttr * sizeof(float));
    hipMemset(d_clusterSizes, 0, k * sizeof(int));

    // Transfer host memory to device memory
    hipMemcpy(d_dataset, h_dataset, numInst * numAttr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, h_centroids, k * numAttr * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(d_clusters, h_clusters, num_inst * sizeof(int), hipMemcpyHostToDevice);

    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int threadsPerBlockForCentroids = 32;
    int gridSizeForDataset = (numInst + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int gridSizeForCentroids = (numAttr * k + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int sharedMemorySize = THREADS_PER_BLOCK * numAttr * sizeof(float) + THREADS_PER_BLOCK * sizeof(int);

    hipError_t hipError_t;
    int iteration = 0;

    // Kmeans
    while(iteration < 150) {
        printf("----------------------- ITERATION %d ---------------------------\n", iteration);

        AssignClusters<<<gridSizeForDataset, THREADS_PER_BLOCK>>>(d_dataset, d_centroids, d_clusters, k, numInst, numAttr);

        hipMemset(d_sumOfCentroids, 0, k * numAttr * sizeof(float));
        hipMemset(d_clusterSizes, 0, k * sizeof(int));

        CalculateClusterSumsAndSizes<<<gridSizeForDataset, THREADS_PER_BLOCK, sharedMemorySize>>>(d_dataset, d_clusters, d_sumOfCentroids, d_clusterSizes, k, numInst, numAttr);

        CalculateCentroidMeans<<<gridSizeForCentroids, threadsPerBlockForCentroids>>>(d_centroids, d_sumOfCentroids, d_clusterSizes, k, numAttr);

        hipError_t = hipGetLastError();
  
        if(hipError_t != hipSuccess) {
            fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
            exit(EXIT_FAILURE);
        }

        iteration++;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Transfer device memory to host memory
    hipMemcpy(h_clusters, d_clusters, numInst * sizeof(int), hipMemcpyDeviceToHost);

    // hipError_t = hipGetLastError();
  
    // if(hipError_t != hipSuccess) {
    //     fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
    //     exit(EXIT_FAILURE);
    // }

    printf("It took %llu ms to process %d datapoints into %d clusters.\n", milliseconds, numInst, k);

    hipFree(d_dataset);
    hipFree(d_centroids);
    hipFree(d_clusters);
    hipFree(d_sumOfCentroids);
    // hipFree(d_centroidDiffs);
    hipFree(d_clusterSizes);
    free(h_dataset);
    free(h_centroids);
    free(h_clusters);
}
